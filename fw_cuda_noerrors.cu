#include "hip/hip_runtime.h"
/* ADOPTED SERIAL CODE IS AS OF NOON 4/22/25 (HILLEL)

   nvcc -arch compute_70 -code sm_70 fw_cuda_tests.cu -o fw_cuda_tests

   qrsh -l gpus=1 -P ec527

EC527 Final Project
Scalar optimizations for Floyd-Warshall Algorithm
- Finds the All-Pairs-Shortest-Paths (APSP) of a randomly generated directed, weighted graph of multiple sizes (represented by adjacency matrices)
- Records the number of cycles taken to run the algorithm for each size

Time measurement code is borrowed from previous EC527 labs.

--------------------------------------------------------------------------------
gcc -O1 fw_scalar_optimizations.c -lrt -o fw_scalar_optimizations

*/

#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <algorithm>


/* =============== Serial Constants =============== */
#define A  32  /* coefficient of x^2 */
#define B  32  /* coefficient of x */
#define C  32  /* constant term */

#define NUM_TESTS 8 // set to 15

#define CPNS 3.0

#define OPTIONS 7

#define IDENT 0

#define INF_EDGE 999999 // arbitrarily large value for infinity edges

#define BLOCK_SIZE 64

typedef int data_t;

/* ============== CUDA Constants =============== */
#define BLOCK_DIM  32   // 32x32 threads per block

#define IDX(i, j, N)    ((i) * (N) + (j))

#define GPU_OPTIONS 6
#define CPU_VERIFICATION 1

/* =================== CUDA Function Prototypes =================== */
void flatten_matrix(int M, int N, int **matrix, int *flat);
void host_FW(int *d, int N);
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
        hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

/* == Naive Kernel: Updates the distance matrix d[i][j] for a fixed k. == */
__global__ void fw_kernel_naive(int *d, int k, int N) {
    // Each thread computes a single element d[i][j] of the distance matrix

    /* int tx = threadIdx.x;        // Thread index in the block
    int ty = threadIdx.y;   // Thread index in the block
    int bx = blockIdx.x;    // Block index in the grid
    int by = blockIdx.y;    // Block index in the grid

    int i = by * blockDim.y + ty;   // Row index
    int j = bx * blockDim.x + tx;   // Column index */
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // Row index
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // Column index

    if (i < N && j < N) {
        // If within bounds of the matrix
        if (d[IDX(i, k, N)] != INF_EDGE && d[IDX(k, j, N)] != INF_EDGE) {
            // If d[i][k] and d[k][j] edges exist
            // d[i][k] + d[k][j] < d[i][j]
            if (d[IDX(i, k, N)] + d[IDX(k, j, N)] < d[IDX(i, j, N)]) {
                // Update distance
                d[IDX(i, j, N)] = d[IDX(i, k, N)] + d[IDX(k, j, N)];
            }
        }
    }
}

/* == Basic Kernel: Updates the distance matrix d[i][j] for a fixed k. == */
__global__ void fw_kernel_basic(int *__restrict__ d, int k, int N) {
    // Each thread computes a single element d[i][j] of the distance matrix

    int i = blockIdx.y * blockDim.y + threadIdx.y;  // Row index
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // Column index

    if (i >= N || j >= N) return; // If out of bounds of matrix, return

    // Precompute indices for the distance matrix
    int idx = i * N + j;

    int sum = d[IDX(i, k, N)] + d[IDX(k, j, N)]; // Sum of distances through k

    /* int sum = dik + dkj; // Sum of distances through k
    // single “conditional move” instead of three nested if’s:
    // if both edges exist AND new path is shorter, pick sum; else keep old dij
    d[idx] = (dik != INF_EDGE && dkj != INF_EDGE && sum < dij) ? sum : dij; */

    // If distance through k is shorter, update distance
    // If either dik or dkj is INF_EDGE, then sum will be > INF_EDGE
    // Don't need to check for INF_EDGE because dij is already < INF_EDGE
    d[idx] = (sum < d[idx]) ? sum : d[idx]; 
}

/* == Basic Min Kernel: Uses min() instead of ternary operator. == */
__global__ void fw_kernel_basic_min(int *__restrict__ d, int k, int N) {
    // Each thread computes a single element d[i][j] of the distance matrix

    int i = blockIdx.y * blockDim.y + threadIdx.y;  // Row index
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // Column index

    if (i >= N || j >= N) return; // If out of bounds of matrix, return

    // Precompute indices for the distance matrix
    int idx = i * N + j;

    int sum = d[IDX(i, k, N)] + d[IDX(k, j, N)]; // Sum of distances through k

    /* int sum = dik + dkj; // Sum of distances through k
    // single “conditional move” instead of three nested if’s:
    // if both edges exist AND new path is shorter, pick sum; else keep old dij
    d[idx] = (dik != INF_EDGE && dkj != INF_EDGE && sum < dij) ? sum : dij; */

    // If distance through k is shorter, update distance
    // If either dik or dkj is INF_EDGE, then sum will be > INF_EDGE
    // Don't need to check for INF_EDGE because dij is already < INF_EDGE
    d[idx] = min(sum, d[idx]); 
}

/* == Blocking Kernel: 3 Phases blocking == */
__global__ void fw_kernel_blocked_allinone(int *__restrict__ d, int k, int N) {
    // Each thread computes a single element d[i][j] of the distance matrix

    // Shared mem for pivot + row + col tiles
    __shared__ int tileK[BLOCK_DIM][BLOCK_DIM]; // pivot tile
    __shared__ int tileI[BLOCK_DIM][BLOCK_DIM]; // row tile
    __shared__ int tileJ[BLOCK_DIM][BLOCK_DIM]; // col tile


    // Block and thread coordinates
    int bi = blockIdx.y; // Block index in the grid (row)
    int bj = blockIdx.x; // Block index in the grid (col)

    int ti = threadIdx.y; // Thread index in the block (row)
    int tj = threadIdx.x; // Thread index in the block (col)


    // Loop over pivot block steps
    for (int kb = 0; kb < N; kb += BLOCK_DIM) {
        
        // Global row and column for pivot block
        int pivot_global_i = kb + ti;
        int pivot_global_j = kb + tj;

        // ---------- Phase 1: Process diagonal (K) tile ----------
        // Load top left pivot tile into shared
        int pivot_block = kb / BLOCK_DIM;

        if (bi == pivot_block && bj == pivot_block) {
            // kb = first element of the pivot block
            tileK[ti][tj] = d[pivot_global_i * N + pivot_global_j];
        }
        __syncthreads();

        // FW on pivot tile, using pivots kb + k2
        if (bi == pivot_block && bj == pivot_block) {
			# pragma unroll
            for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                int via = tileK[ti][k2] + tileK[k2][tj];      // dist through k = d[i][k] + d[k][j]
                //tileK[ti][tj] = v ^ ((tileK[ti][tj] ^ v) & -(tileK[ti][tj] < v));   // min
                tileK[ti][tj] = (via < tileK[ti][tj]) ? via : tileK[ti][tj];
                __syncthreads();
            }
            // Write back updated pivot
            d[pivot_global_i * N + pivot_global_j] = tileK[ti][tj];
        }
        __syncthreads();

        // Load updated pivot tile into shared
        {
            tileK[ti][tj] = d[pivot_global_i * N + pivot_global_j];
        }
        __syncthreads();


        // Global row and column for row blocks
        int row_global_i = kb + ti;
        int row_global_j = bj * BLOCK_DIM + tj;
        // Global row and column for col blocks
        int col_global_i = bi * BLOCK_DIM + ti;
        int col_global_j = kb + tj;

        // ----- Phase 2: Process row (I) and column (J) tiles -----
        // Load pivot row tileI
        if (bi == pivot_block && bj != pivot_block) {
            tileI[ti][tj] = d[row_global_i * N + row_global_j];
        }
        __syncthreads();
        // Load pivot col tileJ
        if (bj == pivot_block && bi != pivot_block) {
            tileJ[ti][tj] = d[col_global_i * N + col_global_j];
        }
        __syncthreads();
        
        // Update row blocks using tileK and tileI
        if (bi == pivot_block && bj != pivot_block) {
            // FW on tileI (col tile), using pivots kb + k2
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                // dist through k = d[i][k] + d[k][j]
                int v = tileK[ti][k2] + tileI[k2][tj];  
                tileI[ti][tj] = (v < tileI[ti][tj]) ? v : tileI[ti][tj];   // min
                __syncthreads();
            }
            // Write back updated row
            d[row_global_i * N + row_global_j] = tileI[ti][tj];
        }
        __syncthreads();
        // Update col blocks using tileK and tileJ
        if (bi != pivot_block && bj == pivot_block) {
            // FW on tileJ (row tile), using pivots kb + k2
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                // dist through k = d[i][k] + d[k][j]
                int v = tileK[k2][tj] + tileJ[ti][k2];
                tileJ[ti][tj] = (v < tileJ[ti][tj]) ? v : tileJ[ti][tj];   // min
                __syncthreads();
            }
            // Write back updated col
            d[col_global_i * N + col_global_j] = tileJ[ti][tj];
        } 
        __syncthreads();
        
        // Load updated pivot row/col tiles into shared in phase 3


        // Global row and column for remaining blocks
        int rem_global_i = bi * BLOCK_DIM + ti;
        int rem_global_j = bj * BLOCK_DIM + tj;

        // ---------- Phase 3: Process remaining off-diagonal tiles ----------
        if (bi != pivot_block && bj != pivot_block) {
            // Load updated tileI and tileJ into shared memory
            tileI[ti][tj] = d[row_global_i * N + row_global_j]; // row tile
            tileJ[ti][tj] = d[col_global_i * N + col_global_j]; // col tile
        }

        // Read current distance into reg
        int myVal = d[rem_global_i * N + rem_global_j]; // current distance
        __syncthreads();

        if (bi != pivot_block && bj != pivot_block) {
            // Relax against all pivots
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                //int v = tileI[ti][k2] + tileJ[k2][tj];
                int v = tileJ[ti][k2] + tileI[k2][tj];
                myVal = v ^ ((myVal ^ v) & -(myVal < v));   // min
                __syncthreads();
            }

            // Write updated value back to global
            d[rem_global_i * N + rem_global_j] = myVal;
        }
        __syncthreads();
    }
    __syncthreads();
}

/* == Blocking Kernel: Reduced __syncthreads() == */
__global__ void fw_kernel_blocked_reduced_sync(int *__restrict__ d, int k, int N) {
    // Each thread computes a single element d[i][j] of the distance matrix

    // Shared mem for pivot + row + col tiles
    __shared__ int tileK[BLOCK_DIM][BLOCK_DIM]; // pivot tile
    __shared__ int tileI[BLOCK_DIM][BLOCK_DIM]; // row tile
    __shared__ int tileJ[BLOCK_DIM][BLOCK_DIM]; // col tile


    // Block and thread coordinates
    int bi = blockIdx.y; // Block index in the grid (row)
    int bj = blockIdx.x; // Block index in the grid (col)

    int ti = threadIdx.y; // Thread index in the block (row)
    int tj = threadIdx.x; // Thread index in the block (col)


    // Loop over pivot block steps
    for (int kb = 0; kb < N; kb += BLOCK_DIM) {
        
        // Global row and column for pivot block
        int pivot_global_i = kb + ti;
        int pivot_global_j = kb + tj;

        // ---------- Phase 1: Process diagonal (K) tile ----------
        // Load top left pivot tile into shared
        int pivot_block = kb / BLOCK_DIM;

        if (bi == pivot_block && bj == pivot_block) {
            // kb = first element of the pivot block
            tileK[ti][tj] = d[pivot_global_i * N + pivot_global_j];
        }
        __syncthreads();

        // FW on pivot tile, using pivots kb + k2
        if (bi == pivot_block && bj == pivot_block) {
			# pragma unroll
            for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                int via = tileK[ti][k2] + tileK[k2][tj];      // dist through k = d[i][k] + d[k][j]
                //tileK[ti][tj] = v ^ ((tileK[ti][tj] ^ v) & -(tileK[ti][tj] < v));   // min
                tileK[ti][tj] = (via < tileK[ti][tj]) ? via : tileK[ti][tj];
                //__syncthreads();
            }
            // Write back updated pivot
            d[pivot_global_i * N + pivot_global_j] = tileK[ti][tj];
        }
        __syncthreads();

        // Load updated pivot tile into shared
        {
            tileK[ti][tj] = d[pivot_global_i * N + pivot_global_j];
        }
        __syncthreads();


        // Global row and column for row blocks
        int row_global_i = kb + ti;
        int row_global_j = bj * BLOCK_DIM + tj;
        // Global row and column for col blocks
        int col_global_i = bi * BLOCK_DIM + ti;
        int col_global_j = kb + tj;

        // ----- Phase 2: Process row (I) and column (J) tiles -----
        // Load pivot row tileI
        if (bi == pivot_block && bj != pivot_block) {
            tileI[ti][tj] = d[row_global_i * N + row_global_j];
        }
        //__syncthreads();
        // Load pivot col tileJ
        if (bj == pivot_block && bi != pivot_block) {
            tileJ[ti][tj] = d[col_global_i * N + col_global_j];
        }
        __syncthreads();
        
        // Update row blocks using tileK and tileI
        if (bi == pivot_block && bj != pivot_block) {
            // FW on tileI (col tile), using pivots kb + k2
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                // dist through k = d[i][k] + d[k][j]
                int v = tileK[ti][k2] + tileI[k2][tj];  
                tileI[ti][tj] = (v < tileI[ti][tj]) ? v : tileI[ti][tj];   // min
                //__syncthreads();
            }
            // Write back updated row
            d[row_global_i * N + row_global_j] = tileI[ti][tj];
        }
        //__syncthreads();
        // Update col blocks using tileK and tileJ
        if (bi != pivot_block && bj == pivot_block) {
            // FW on tileJ (row tile), using pivots kb + k2
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                // dist through k = d[i][k] + d[k][j]
                int v = tileK[k2][tj] + tileJ[ti][k2];
                tileJ[ti][tj] = (v < tileJ[ti][tj]) ? v : tileJ[ti][tj];   // min
                //__syncthreads();
            }
            // Write back updated col
            d[col_global_i * N + col_global_j] = tileJ[ti][tj];
        } 
        __syncthreads();
        
        // Load updated pivot row/col tiles into shared in phase 3


        // Global row and column for remaining blocks
        int rem_global_i = bi * BLOCK_DIM + ti;
        int rem_global_j = bj * BLOCK_DIM + tj;

        // ---------- Phase 3: Process remaining off-diagonal tiles ----------
        if (bi != pivot_block && bj != pivot_block) {
            // Load updated tileI and tileJ into shared memory
            tileI[ti][tj] = d[row_global_i * N + row_global_j]; // row tile
            tileJ[ti][tj] = d[col_global_i * N + col_global_j]; // col tile
        }

        // Read current distance into reg
        int myVal = d[rem_global_i * N + rem_global_j]; // current distance
        __syncthreads();

        if (bi != pivot_block && bj != pivot_block) {
            // Relax against all pivots
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                //int v = tileI[ti][k2] + tileJ[k2][tj];
                int v = tileJ[ti][k2] + tileI[k2][tj];
                myVal = v ^ ((myVal ^ v) & -(myVal < v));   // min
                //__syncthreads();
            }

            // Write updated value back to global
            d[rem_global_i * N + rem_global_j] = myVal;
        }
        __syncthreads();
    }
    __syncthreads();
}

/* == Blocking Kernel: Tile padding for memory banks == */
__global__ void fw_kernel_blocked_padded(int *__restrict__ d, int k, int N) {
    // Each thread computes a single element d[i][j] of the distance matrix

    // Shared mem for pivot + row + col tiles
    __shared__ int tileK[BLOCK_DIM + 1][BLOCK_DIM + 1]; // pivot tile
    __shared__ int tileI[BLOCK_DIM + 1][BLOCK_DIM + 1]; // row tile
    __shared__ int tileJ[BLOCK_DIM + 1][BLOCK_DIM + 1]; // col tile


    // Block and thread coordinates
    int bi = blockIdx.y; // Block index in the grid (row)
    int bj = blockIdx.x; // Block index in the grid (col)

    int ti = threadIdx.y; // Thread index in the block (row)
    int tj = threadIdx.x; // Thread index in the block (col)


    // Loop over pivot block steps
    for (int kb = 0; kb < N; kb += BLOCK_DIM) {
        
        // Global row and column for pivot block
        int pivot_global_i = kb + ti;
        int pivot_global_j = kb + tj;

        // ---------- Phase 1: Process diagonal (K) tile ----------
        // Load top left pivot tile into shared
        int pivot_block = kb / BLOCK_DIM;

        if (bi == pivot_block && bj == pivot_block) {
            // kb = first element of the pivot block
            tileK[ti][tj] = d[pivot_global_i * N + pivot_global_j];
        }
        __syncthreads();

        // FW on pivot tile, using pivots kb + k2
        if (bi == pivot_block && bj == pivot_block) {
			# pragma unroll
            for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                int via = tileK[ti][k2] + tileK[k2][tj];      // dist through k = d[i][k] + d[k][j]
                //tileK[ti][tj] = v ^ ((tileK[ti][tj] ^ v) & -(tileK[ti][tj] < v));   // min
                tileK[ti][tj] = (via < tileK[ti][tj]) ? via : tileK[ti][tj];
                __syncthreads();
            }
            // Write back updated pivot
            d[pivot_global_i * N + pivot_global_j] = tileK[ti][tj];
        }
        __syncthreads();

        // Load updated pivot tile into shared
        {
            tileK[ti][tj] = d[pivot_global_i * N + pivot_global_j];
        }
        __syncthreads();


        // Global row and column for row blocks
        int row_global_i = kb + ti;
        int row_global_j = bj * BLOCK_DIM + tj;
        // Global row and column for col blocks
        int col_global_i = bi * BLOCK_DIM + ti;
        int col_global_j = kb + tj;

        // ----- Phase 2: Process row (I) and column (J) tiles -----
        // Load pivot row tileI
        if (bi == pivot_block && bj != pivot_block) {
            tileI[ti][tj] = d[row_global_i * N + row_global_j];
        }
        __syncthreads();
        // Load pivot col tileJ
        if (bj == pivot_block && bi != pivot_block) {
            tileJ[ti][tj] = d[col_global_i * N + col_global_j];
        }
        __syncthreads();
        
        // Update row blocks using tileK and tileI
        if (bi == pivot_block && bj != pivot_block) {
            // FW on tileI (col tile), using pivots kb + k2
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                // dist through k = d[i][k] + d[k][j]
                int v = tileK[ti][k2] + tileI[k2][tj];  
                tileI[ti][tj] = (v < tileI[ti][tj]) ? v : tileI[ti][tj];   // min
                __syncthreads();
            }
            // Write back updated row
            d[row_global_i * N + row_global_j] = tileI[ti][tj];
        }
        __syncthreads();
        // Update col blocks using tileK and tileJ
        if (bi != pivot_block && bj == pivot_block) {
            // FW on tileJ (row tile), using pivots kb + k2
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                // dist through k = d[i][k] + d[k][j]
                int v = tileK[k2][tj] + tileJ[ti][k2];
                tileJ[ti][tj] = (v < tileJ[ti][tj]) ? v : tileJ[ti][tj];   // min
                __syncthreads();
            }
            // Write back updated col
            d[col_global_i * N + col_global_j] = tileJ[ti][tj];
        } 
        __syncthreads();
        
        // Load updated pivot row/col tiles into shared in phase 3


        // Global row and column for remaining blocks
        int rem_global_i = bi * BLOCK_DIM + ti;
        int rem_global_j = bj * BLOCK_DIM + tj;

        // ---------- Phase 3: Process remaining off-diagonal tiles ----------
        if (bi != pivot_block && bj != pivot_block) {
            // Load updated tileI and tileJ into shared memory
            tileI[ti][tj] = d[row_global_i * N + row_global_j]; // row tile
            tileJ[ti][tj] = d[col_global_i * N + col_global_j]; // col tile
        }

        // Read current distance into reg
        int myVal = d[rem_global_i * N + rem_global_j]; // current distance
        __syncthreads();

        if (bi != pivot_block && bj != pivot_block) {
            // Relax against all pivots
            # pragma unroll
			for (int k2 = 0; k2 < BLOCK_DIM; k2++) {
                //int v = tileI[ti][k2] + tileJ[k2][tj];
                int v = tileJ[ti][k2] + tileI[k2][tj];
                myVal = v ^ ((myVal ^ v) & -(myVal < v));   // min
                __syncthreads();
            }

            // Write updated value back to global
            d[rem_global_i * N + rem_global_j] = myVal;
        }
        __syncthreads();
    }
    __syncthreads();
}

/* =================== Serial Function Prototypes =================== */
int clock_gettime(clockid_t clk_id, struct timespec *tp);
int **create_adjacency_matrix(int num_vertices);
void free_adjacency_matrix(int **matrix, int num_vertices);
void print_graph(int **graph, int num_vertices);
void fw_serial(int **graph, int num_vertices);
void fw_local_variables(int **graph, int num_vertices);
void fw_conditional_move(int **graph, int num_vertices);
void fw_loop_unroll2(int **graph, int num_vertices);
void fw_loop_unroll4(int **graph, int num_vertices);
void fw_loop_unroll8(int **graph, int num_vertices);
void fw_loop_unroll4_lvars(int **graph, int num_vertices);
void fw_blocked(int **graph, int num_vertices);
void process_block(int **graph, int num_vertices, int i, int j, int k);
void process_block_lvars(int **graph, int num_vertices, int i, int j, int k);
void process_block_unroll4(int **graph, int num_vertices, int i, int j, int k);

/* =================== Serial Time Measurement =================== */

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

/* =================== Serial Wakeup Delay =================== */

double wakeup_delay()
{
  double meas = 0; int i, j;
  struct timespec time_start, time_stop;
  double quasi_random = 0;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
  j = 100;
  while (meas < 1.0) {
    for (i = 1; i < j; i++) {
      /* This iterative calculation uses a chaotic map function, specifically
         the complex quadratic map (as in Julia and Mandelbrot sets), which is
         unpredictable enough to prevent compiler optimisation. */
      quasi_random = quasi_random*quasi_random - 1.923432;
    }
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
    meas = interval(time_start, time_stop);
    j *= 2; /* Twice as much delay next time, until we've taken 1 second */
  }
  return quasi_random;
}

/* =================== Main Function =================== */
void fw_CPU() {
    int OPTION;
    int num_vertices, max_vertices;
    struct timespec time_start, time_stop;
    double time_stamp[OPTIONS][NUM_TESTS];
    double wd;
    int **graph;
    int **ref_graph[NUM_TESTS];
    int x;

    printf("Floyd-Warshall Algorithm - Serial Implementation\n");

    wd = wakeup_delay();

    x = NUM_TESTS - 1;
    max_vertices = A*x*x + B*x + C;

    for (OPTION = 0; OPTION < OPTIONS; OPTION++) {
        printf("Testing option %d\n", OPTION);
        for (x = 0; x < NUM_TESTS && (num_vertices = A*x*x + B*x + C, num_vertices <= max_vertices); x++) {

            // create the adjacency matrix
            graph = create_adjacency_matrix(num_vertices);

            // start timing the algorithm
            clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);

            switch(OPTION) {
                case 0: // serial/baseline implementation
                    fw_serial(graph, num_vertices);
                    break;
                case 1: // using local variables
                    fw_local_variables(graph, num_vertices); // local variables implementation
                    break;
                case 2: // unrolled implementation by factor of 2
                    fw_loop_unroll2(graph, num_vertices);
                    break;
                case 3: // unrolled implementation by factor of 4
                    fw_loop_unroll4(graph, num_vertices);
                    break;
                case 4: // unrolled implementation by factor of 8
                    fw_loop_unroll8(graph, num_vertices);
                    break;
                case 5: // unrolled implementation by factor of 4 with local variables
                    fw_loop_unroll4_lvars(graph, num_vertices);
                    break;
                case 6: // blocked implementation
                    fw_blocked(graph, num_vertices);
                    break;
                default:
                    break;
            }

            // stop timing the algorithm
            clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);

            // calculate and store the time taken
            time_stamp[OPTION][x] = interval(time_start, time_stop);

            // copy resulting graph to reference graph for comparison
            if (OPTION == 0) {
                ref_graph[x] = (int **)malloc(num_vertices * sizeof(int *));
                for (int i = 0; i < num_vertices; i++) {
                    ref_graph[x][i] = (int *)malloc(num_vertices * sizeof(int));
                    for (int j = 0; j < num_vertices; j++) {
                        ref_graph[x][i][j] = graph[i][j];
                    }
                }
            }

            // check if the results are correct (based on results from serial implementation)
            if (OPTION != 0) {
                for (int i = 0; i < num_vertices; i++) {
                    for (int j = 0; j < num_vertices; j++) {
                        if (graph[i][j] != ref_graph[x][i][j]) {
                            printf("Error: Results do not match for option %d at (%d, %d)\n", OPTION, i, j);
                            time_stamp[OPTION][x] = 0;
                            break;
                        }
                    }
                }
            }

            printf("  iter %d done\r", x); fflush(stdout);

            // Free the adjacency matrix memory
            free_adjacency_matrix(graph, num_vertices);
        }
    }

    printf("\nnum_vertices, baseline, local variables, unroll 2x, unroll 4x, unroll 8x, unroll 4x with local vars, blocked \n");
    for (x = 0; x < NUM_TESTS && (num_vertices = A*x*x + B*x + C, num_vertices <= max_vertices); x++) {
        printf("%d", num_vertices);
        for (OPTION = 0; OPTION < OPTIONS; OPTION++) {
            printf(", %ld", (long int)((double)(CPNS) * 1.0e9 * time_stamp[OPTION][x]));
        }
        printf("\n");
    }

    printf("\n");
    printf("Initial delay was calculating: %g \n", wd);

}

void fw_GPU() {
    int OPTION;
    int num_vertices, max_vertices;
    int **graph;
    float time_stamp_GPU_data[GPU_OPTIONS][NUM_TESTS];
    float time_stamp_GPU_calc[GPU_OPTIONS][NUM_TESTS];
    int x;

    // GPU Timing Variables
    hipEvent_t startData, endData, startFW, endFW;
    float elapsedGPUData, elapsedGPUFW;

    // Arrays on GPU global memory
    int *d_d;       // Distance matrix on GPU
    // Arrays on host memory
    int *h_d;       // Distance matrix on host
    int *h_d_gold;  // Gold distance matrix for verification

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    // Create CUDA events for timing
    CUDA_SAFE_CALL(hipEventCreate(&startData));
    CUDA_SAFE_CALL(hipEventCreate(&endData));
    CUDA_SAFE_CALL(hipEventCreate(&startFW));
    CUDA_SAFE_CALL(hipEventCreate(&endFW));

    printf("\n\n\nFloyd-Warshall Algorithm - GPU Implementation\n\n");

    x = NUM_TESTS - 1;
    max_vertices = A*x*x + B*x + C;

    for (OPTION = 0; OPTION < GPU_OPTIONS; OPTION++) {
        printf("Testing GPU option %d\n", OPTION);
        for (x = 0; x < NUM_TESTS && (num_vertices = A*x*x + B*x + C, num_vertices <= max_vertices); x++) {
            int N = num_vertices;

            // Allocate GPU memory
            size_t allocSize = N*N * sizeof(int);
            CUDA_SAFE_CALL(hipMalloc((void**)&d_d, allocSize));
            // Allocate host memory
            h_d = (int *)malloc(allocSize);
            h_d_gold = (int *)malloc(allocSize);

            // create the adjacency matrix
            graph = create_adjacency_matrix(num_vertices);
            // Initialize host arrays
            flatten_matrix(N, N, graph, h_d);
            flatten_matrix(N, N, graph, h_d_gold);

            // Record start event for data transfer
            CUDA_SAFE_CALL(hipEventRecord(startData, 0));

            // Transfer arrays to GPU memory
            CUDA_SAFE_CALL(hipMemcpy(d_d, h_d, allocSize, hipMemcpyHostToDevice));

            // Record start event for Floyd-Warshall kernel
            CUDA_SAFE_CALL(hipEventRecord(startFW, 0));

            switch(OPTION) {
                case 0: { // naive GPU implementation
                    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
                    dim3 dimGrid( (N + BLOCK_DIM - 1) / BLOCK_DIM,
                                  (N + BLOCK_DIM - 1) / BLOCK_DIM );
                    for (int k = 0; k < N; k++) {
                        // Launch kernel for each k iteration
                        fw_kernel_naive<<<dimGrid, dimBlock>>>(d_d, k, N);
                        CUDA_SAFE_CALL(hipGetLastError());
                        CUDA_SAFE_CALL(hipDeviceSynchronize());
                    }
                    break;
                }
				case 1: { // basic GPU implementation
                    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
                    dim3 dimGrid( (N + BLOCK_DIM - 1) / BLOCK_DIM,
                                  (N + BLOCK_DIM - 1) / BLOCK_DIM );
                    for (int k = 0; k < N; k++) {
                        // Launch kernel for each k iteration
                        fw_kernel_basic<<<dimGrid, dimBlock>>>(d_d, k, N);
                        CUDA_SAFE_CALL(hipGetLastError());
                    }
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
                    break;
                }
				case 2: { // basic min GPU implementation
                    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
                    dim3 dimGrid( (N + BLOCK_DIM - 1) / BLOCK_DIM,
                                  (N + BLOCK_DIM - 1) / BLOCK_DIM );
                    for (int k = 0; k < N; k++) {
                        // Launch kernel for each k iteration
                        fw_kernel_basic_min<<<dimGrid, dimBlock>>>(d_d, k, N);
                        CUDA_SAFE_CALL(hipGetLastError());
                    }
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
                    break;
                }
                case 3: {   // GPU blocked all in one
                    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
                    dim3 dimGrid( (N + BLOCK_DIM - 1) / BLOCK_DIM,
                                  (N + BLOCK_DIM - 1) / BLOCK_DIM );
                    //printf("Launching AIO Blocked Kernel...\n");
                    fw_kernel_blocked_allinone<<<dimGrid, dimBlock>>>(d_d, 0, N);
                    CUDA_SAFE_CALL(hipGetLastError());
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
                }
				case 4: {   // GPU blocked reduced __syncthreads()
                    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
                    dim3 dimGrid( (N + BLOCK_DIM - 1) / BLOCK_DIM,
                                  (N + BLOCK_DIM - 1) / BLOCK_DIM );
                    //printf("Launching AIO Blocked Kernel...\n");
                    fw_kernel_blocked_reduced_sync<<<dimGrid, dimBlock>>>(d_d, 0, N);
                    CUDA_SAFE_CALL(hipGetLastError());
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
                }
				case 5: {   // GPU blocked padded tiles
                    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
                    dim3 dimGrid( (N + BLOCK_DIM - 1) / BLOCK_DIM,
                                  (N + BLOCK_DIM - 1) / BLOCK_DIM );
                    //printf("Launching AIO Blocked Kernel...\n");
                    fw_kernel_blocked_padded<<<dimGrid, dimBlock>>>(d_d, 0, N);
                    CUDA_SAFE_CALL(hipGetLastError());
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
                }
                default:
                    break;
            }

            // Record end event for Floyd-Warshall kernel
            CUDA_SAFE_CALL(hipDeviceSynchronize());
            CUDA_SAFE_CALL(hipEventRecord(endFW, 0));

            // Transfer results back to host
            CUDA_SAFE_CALL(hipMemcpy(h_d, d_d, allocSize, hipMemcpyDeviceToHost));

            // Record end event for data transfer
            CUDA_SAFE_CALL(hipEventRecord(endData, 0));

            // Stop timers
            CUDA_SAFE_CALL(hipEventSynchronize(endFW));
            CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedGPUFW, startFW, endFW));
            CUDA_SAFE_CALL(hipEventSynchronize(endData));
            CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedGPUData, startData, endData));

            //printf("\nGPU Time (w/ data transfer): %f ms\n", elapsedGPUData);
            //printf("GPU Time (kernel only): %f ms\n", elapsedGPUFW);
            // Calculate and store time taken
            time_stamp_GPU_data[OPTION][x] = elapsedGPUData;
            time_stamp_GPU_calc[OPTION][x] = elapsedGPUFW;


            // Verify GPU results
			if (CPU_VERIFICATION) {
				host_FW_unroll4(h_d_gold, N);
				int errCount = 0;
				int max_diff = 0;
				//printf("GPU, CPU\n");
				for (int i = 0; i < N*N; i++) {
					float diff = abs(h_d[i] - h_d_gold[i]);
					if (diff > 1) errCount++;
					if (diff > max_diff) max_diff = diff;
					
					//printf("(%d,%d) ", h_d[i], h_d_gold[i]);
					//if (i % N == N - 1) printf("\n");
				}
				if (errCount > 0) {
					printf("\n        ERROR: %d elements do not match\n", errCount);
					printf("        Max difference between CPU and GPU results: %d\n", max_diff);
				} else {
					//printf("\nTEST PASSED: All elements match\n");
				}
			}

            // Free device and host memory
            CUDA_SAFE_CALL(hipFree(d_d));
            free(h_d);
            free(h_d_gold);
            free_adjacency_matrix(graph, num_vertices);

            int gridDim = (N + BLOCK_DIM - 1) / BLOCK_DIM;
            printf("  iter %d done with %dx%d grid\r", x, gridDim, gridDim); fflush(stdout);
        }
    }

    printf("\nGPU Time (ms): Calculation Only / Incl. Data Transfers\nnum_vertices, Naive GPU (kern), Naive GPU (data), basic (kern), basic (data), min (kern), min (data), blocked (kern), blocked (data), reduced sync (kern), reduced sync (data), padded tiles (kern), padded tiles (data)\n");
    for (x = 0; x < NUM_TESTS && (num_vertices = A*x*x + B*x + C, num_vertices <= max_vertices); x++) {
        printf("%d", num_vertices);
        for (OPTION = 0; OPTION < GPU_OPTIONS; OPTION++) {
            printf(", %f, %f", time_stamp_GPU_calc[OPTION][x], time_stamp_GPU_data[OPTION][x]);
        }
        printf("\n");
    }

    // Destroy CUDA timers
    CUDA_SAFE_CALL(hipEventDestroy(startData));
    CUDA_SAFE_CALL(hipEventDestroy(endData));
    CUDA_SAFE_CALL(hipEventDestroy(startFW));
    CUDA_SAFE_CALL(hipEventDestroy(endFW));
}

int main() {

    //fw_CPU();
    fw_GPU();

    return 0;
}


/* =================== CUDA Function Definitions =================== */
void flatten_matrix(int M, int N, int **matrix, int *flat) {
        int i, j;
        for (i = 0; i < M; i++) {
                for (j = 0; j < N; j++) {
                        flat[i * N + j] = matrix[i][j];
                }
        }
}

void host_FW(int *d, int N) {
    // Host implementation of Floyd-Warshall algorithm
    for (int k = 0; k < N; k++) {
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                if (d[IDX(i, k, N)] != INF_EDGE && d[IDX(k, j, N)] != INF_EDGE) {
                    int dik = d[IDX(i, k, N)];      // d[i][k]
                    int dkj = d[IDX(k, j, N)];      // d[k][j]
                    int dij = d[IDX(i, j, N)];      // d[i][j]

                    if (dik + dkj < dij) {
                        d[IDX(i, j, N)] = dik + dkj;        // Update distance
                    }
                }
            }
        }
    }
}

void host_FW_unroll4(int *graph, int num_vertices) {
	int i, j, k, ik;
    // unroll the innermost loop by a factor of 4 with local variables
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            // index of the (i,k) element in the flattened array
            ik = graph[i * num_vertices + k];
            // process j in strides of 4
            for (j = 0; j < num_vertices; j += 4) {
                int base_ij = i * num_vertices + j;     // index of (i, j)
                int base_kj = k * num_vertices + j;     // index of (k, j)

                // compute sums
                int sum1 = ik + graph[base_kj];
                int sum2 = ik + graph[base_kj + 1];
                int sum3 = ik + graph[base_kj + 2];
                int sum4 = ik + graph[base_kj + 3];

                // compare/update
                if (sum1 < graph[base_ij]) {
                    graph[base_ij] = sum1;
                }
                if (j + 1 < num_vertices && sum2 < graph[base_ij + 1]) {
                    graph[base_ij + 1] = sum2;
                }
                if (j + 2 < num_vertices && sum3 < graph[base_ij + 2]) {
                    graph[base_ij + 2] = sum3;
                }
                if (j + 3 < num_vertices && sum4 < graph[base_ij + 3]) {
                    graph[base_ij + 3] = sum4;
                }
            }
        }
    }
}


/* =================== Serial Function Definitions =================== */
void fw_serial(int **graph, int num_vertices) {
    int i, j, k;
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            for (j = 0; j < num_vertices; j++) {
                if (graph[i][k] + graph[k][j] < graph[i][j]) {
                    graph[i][j] = graph[i][k] + graph[k][j];
                }
            }
        }
    }
}

void fw_conditional_move(int **graph, int num_vertices) {
    int i, j, k, sum;
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            for (j = 0; j < num_vertices; j++) {
                sum = graph[i][k] + graph[k][j];
                // user ternary operator to conditionally move the value
                graph[i][j] = (sum < graph[i][j]) ? sum : graph[i][j];
            }
        }
    }
}

void fw_local_variables(int **graph, int num_vertices) {
    int i, j, k;
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            int ik = graph[i][k];
            for (j = 0; j < num_vertices; j++) {
                int sum = ik + graph[k][j];
                if (sum < graph[i][j]) {
                    graph[i][j] = sum;
                }
            }
        }
    }
}

void fw_loop_unroll2(int **graph, int num_vertices) {
    int i, j, k;
    // unroll the innermost loop by a factor of 2
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            for (j = 0; j < num_vertices; j += 2) {
                if (graph[i][k] + graph[k][j] < graph[i][j]) {
                    graph[i][j] = graph[i][k] + graph[k][j];
                }
                if (j + 1 < num_vertices && graph[i][k] + graph[k][j + 1] < graph[i][j + 1]) {
                    graph[i][j + 1] = graph[i][k] + graph[k][j + 1];
                }
            }
        }
    }
}

void fw_loop_unroll4(int **graph, int num_vertices) {
    int i, j, k;
    // unroll the innermost loop by a factor of 4
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            for (j = 0; j < num_vertices; j += 4) {
                if (graph[i][k] + graph[k][j] < graph[i][j]) {
                    graph[i][j] = graph[i][k] + graph[k][j];
                }
                if (j + 1 < num_vertices && graph[i][k] + graph[k][j + 1] < graph[i][j + 1]) {
                    graph[i][j + 1] = graph[i][k] + graph[k][j + 1];
                }
                if (j + 2 < num_vertices && graph[i][k] + graph[k][j + 2] < graph[i][j + 2]) {
                    graph[i][j + 2] = graph[i][k] + graph[k][j + 2];
                }
                if (j + 3 < num_vertices && graph[i][k] + graph[k][j + 3] < graph[i][j + 3]) {
                    graph[i][j + 3] = graph[i][k] + graph[k][j + 3];
                }
            }
        }
    }
}

void fw_loop_unroll8(int **graph, int num_vertices) {
    int i, j, k;
    // unroll the innermost loop by a factor of 8
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            for (j = 0; j < num_vertices; j += 8) {
                if (graph[i][k] + graph[k][j] < graph[i][j]) {
                    graph[i][j] = graph[i][k] + graph[k][j];
                }
                if (j + 1 < num_vertices && graph[i][k] + graph[k][j + 1] < graph[i][j + 1]) {
                    graph[i][j + 1] = graph[i][k] + graph[k][j + 1];
                }
                if (j + 2 < num_vertices && graph[i][k] + graph[k][j + 2] < graph[i][j + 2]) {
                    graph[i][j + 2] = graph[i][k] + graph[k][j + 2];
                }
                if (j + 3 < num_vertices && graph[i][k] + graph[k][j + 3] < graph[i][j + 3]) {
                    graph[i][j + 3] = graph[i][k] + graph[k][j + 3];
                }
                if (j + 4 < num_vertices && graph[i][k] + graph[k][j + 4] < graph[i][j + 4]) {
                    graph[i][j + 4] = graph[i][k] + graph[k][j + 4];
                }
                if (j + 5 < num_vertices && graph[i][k] + graph[k][j + 5] < graph[i][j + 5]) {
                    graph[i][j + 5] = graph[i][k] + graph[k][j + 5];
                }
                if (j + 6 < num_vertices && graph[i][k] + graph[k][j + 6] < graph[i][j + 6]) {
                    graph[i][j + 6] = graph[i][k] + graph[k][j + 6];
                }
                if (j + 7 < num_vertices && graph[i][k] + graph[k][j + 7] < graph[i][j + 7]) {
                    graph[i][j + 7] = graph[i][k] + graph[k][j + 7];
                }
            }
        }
    }
}

void fw_loop_unroll4_lvars(int **graph, int num_vertices) {
    int i, j, k, ik;
    // unroll the innermost loop by a factor of 4 with local variables
    for (k = 0; k < num_vertices; k++) {
        for (i = 0; i < num_vertices; i++) {
            ik = graph[i][k];
            for (j = 0; j < num_vertices; j += 4) {
                int sum1 = ik + graph[k][j];
                int sum2 = ik + graph[k][j + 1];
                int sum3 = ik + graph[k][j + 2];
                int sum4 = ik + graph[k][j + 3];
                if (sum1 < graph[i][j]) {
                    graph[i][j] = sum1;
                }
                if (j + 1 < num_vertices && sum2 < graph[i][j + 1]) {
                    graph[i][j + 1] = sum2;
                }
                if (j + 2 < num_vertices && sum3 < graph[i][j + 2]) {
                    graph[i][j + 2] = sum3;
                }
                if (j + 3 < num_vertices && sum4 < graph[i][j + 3]) {
                    graph[i][j + 3] = sum4;
                }
            }
        }
    }
}

void fw_blocked(int **graph, int num_vertices) {
    int i, j, k;
    for (k = 0; k < num_vertices; k += BLOCK_SIZE) {
        // process the diagonal block
        process_block_lvars(graph, num_vertices, k, k, k);

        // process row and column blocks
        for (i = 0; i < num_vertices; i += BLOCK_SIZE) {
            if (i != k) {
                process_block_lvars(graph, num_vertices, i, k, k);
                process_block_lvars(graph, num_vertices, k, i, k);
            }
        }

        // process the remaining blocks
        for (i = 0; i < num_vertices; i += BLOCK_SIZE) {
            if (i == k) continue;
            for (j = 0; j < num_vertices; j += BLOCK_SIZE) {
                if (j == k) continue;
                process_block_lvars(graph, num_vertices, i, j, k);
            }
        }
    }
}

void process_block(int **graph, int num_vertices, int i, int j, int k) {
    for (int kk = k; kk < k + BLOCK_SIZE && kk < num_vertices; kk++) {
        for (int ii = i; ii < i + BLOCK_SIZE && ii < num_vertices; ii++) {
            for (int jj = j; jj < j + BLOCK_SIZE && jj < num_vertices; jj++) {
                if (graph[ii][kk] + graph[kk][jj] < graph[ii][jj]) {
                    graph[ii][jj] = graph[ii][kk] + graph[kk][jj];
                }
            }
        }
    }
}

void process_block_lvars(int **graph, int num_vertices, int i, int j, int k) {
    for (int kk = k; kk < k + BLOCK_SIZE && kk < num_vertices; kk++) {
        for (int ii = i; ii < i + BLOCK_SIZE && ii < num_vertices; ii++) {
            int ik = graph[ii][kk];
            for (int jj = j; jj < j + BLOCK_SIZE && jj < num_vertices; jj++) {
                int sum = ik + graph[kk][jj];
                if (sum < graph[ii][jj]) {
                    graph[ii][jj] = sum;
                }
            }
        }
    }
}

// Create an adjacency matrix for the graph
// Randomly generate edges with weights between 1 and 10
// Set the diagonal to 0 and non-edges to a large value (infinity)
int **create_adjacency_matrix(int num_vertices) {
    srand(2468); // set seed for reproducibility
    int **matrix = (int **)malloc(num_vertices * sizeof(int *));
    for (int i = 0; i < num_vertices; i++) {
        matrix[i] = (int *)malloc(num_vertices * sizeof(int));
        for (int j = 0; j < num_vertices; j++) {
            if (i == j) {
                matrix[i][j] = 0;
            } else {
                // let there be a 70% chance of having an edge
                if ((rand() % 100) < 70) {
                    matrix[i][j] = rand() % 10 + 1; // random weight of 1-10
                } else {
                    matrix[i][j] = INF_EDGE; // no edge, set to "infinity"
                }
            }
        }
    }
    return matrix;
}

// Free the adjacency matrix memory
void free_adjacency_matrix(int **matrix, int num_vertices) {
    for (int i = 0; i < num_vertices; i++) {
        free(matrix[i]);
    }
    free(matrix);
}

// Print the adjacency matrix (for debugging purposes)
void print_graph(int **graph, int num_vertices) {
    for (int i = 0; i < num_vertices; i++) {
        for (int j = 0; j < num_vertices; j++) {
            if (graph[i][j] == INF_EDGE) {
                printf("INF ");
            } else {
                printf("%d ", graph[i][j]);
            }
        }
        printf("\n");
    }
}